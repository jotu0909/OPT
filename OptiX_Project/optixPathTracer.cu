#include "hip/hip_runtime.h"
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include "optixPathTracer.h"
#include "include\random.h"
#include "include\helpers.h"
#include <optix_device.h>

using namespace optix;

struct PerRayData_pathtrace
{
	float3 result;
	float3 normal;
	float3 radiance;
	float3 attenuation;
	float3 origin;
	float3 direction;
	unsigned int seed;
	int depth;
	int countEmitted;
	int done;
	float depthMap;
	bool alpha;

};

struct PerRayData_pathtrace_shadow
{
	bool inShadow;
};

// Scene wide variables
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );


//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );

rtDeclareVariable(unsigned int, frame_number, , );
rtDeclareVariable(unsigned int, sqrt_num_samples, , );
//rtDeclareVariable(float, sqrt_num_samples, , );
rtDeclareVariable(unsigned int, rr_begin_depth, , );
rtDeclareVariable(unsigned int, pathtrace_ray_type, , );
rtDeclareVariable(unsigned int, pathtrace_shadow_ray_type, , );


rtDeclareVariable(float3, emission_color, , );
rtDeclareVariable(float3, diffuse_color, , );
rtDeclareVariable(float3, specular_color, , );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );

rtDeclareVariable(float3, bg_color, , );
rtDeclareVariable(float3, bad_color, , );

rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(unsigned int, trace_depth, , );


rtTextureSampler<float4, 2> Kd_map;
rtTextureSampler<float4, 2> d_map;
rtTextureSampler<float4, 2> Ks_map;
rtTextureSampler<float4, 2> bump_map;

rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(Matrix3x3, normal_matrix, , );

rtBuffer<float4, 2>              output_buffer;
rtBuffer<float4, 2>			     input_normal_buffer;
rtBuffer<float4, 2>			     input_depth_buffer;
rtBuffer<ParallelogramLight>     lights;

rtDeclareVariable(int, lerp_bol, , );


RT_PROGRAM void pathtrace_camera()
{
	//printf("%f, %f, %f\n", Ka.x, Ka.y, Ka.z);
		size_t2 screen = output_buffer.size();

		float2 inv_screen = 1.0f / make_float2(screen) * 2.f;
		float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

		float2 jitter_scale = inv_screen / sqrt_num_samples;

		unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;
		float3 result = make_float3(0.0f);
		float3 normal = make_float3(0.0f);
		float depth_map = 0.0f;
		//bool alpha = false;

		unsigned int seed = tea<16>(screen.x*launch_index.y + launch_index.x, frame_number);

		do
		{
			//
			// Sample pixel using jittering
			//
			unsigned int x = samples_per_pixel%sqrt_num_samples;
			//unsigned int x = 0;
			unsigned int y = samples_per_pixel / sqrt_num_samples;
			float2 jitter = make_float2(x /*- rnd(seed)*/, y /*- rnd(seed)*/);
			float2 d = pixel + jitter*jitter_scale;
			float3 ray_origin = eye;
			float3 ray_direction = normalize(d.x*U + d.y*V + W);

			// Initialze per-ray data
			PerRayData_pathtrace prd;
			prd.result = make_float3(0.f);
			prd.attenuation = make_float3(1.f);
			prd.countEmitted = true;
			prd.done = false;
			prd.seed = seed;
			prd.depth = 0;
			prd.depthMap = 0.0f;
			prd.normal = make_float3(0.0);
			prd.alpha = false;

			// Each iteration is a segment of the ray path.  The closest hit will
			// return new segments to be traced here.
			for (;;)
			{
				Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
				rtTrace(top_object, ray, prd);
				
				if (prd.done)
				{
					// We have hit the background or a luminaire
						prd.result += prd.radiance * prd.attenuation; 
					break;
				}
				
				// Russian roulette termination 
				if (prd.depth >= rr_begin_depth)//
				{
					//if(prd.depth>trace_depth-1){
					//	//printf("%d\n",prd.depth);
					//	prd.result += prd.radiance * prd.attenuation;
					//	break;
					//	//prd.done = true;
					//}
					float pcont = fmaxf(prd.attenuation);
					if (rnd(prd.seed) >= pcont)
						break;
					prd.attenuation /= pcont;			
				}
					
				//if (prd.alpha) {
				//	prd.depth++;
				//	prd.alpha = false;
				//}


					prd.depth++;
				
				//// Without Russian roulette termination 
				//prd.done = true;
				
				prd.result += prd.radiance * prd.attenuation;

				// Update ray data for the next path segment
				ray_origin = prd.origin;
				ray_direction = prd.direction;
			}

			result += prd.result;
			float3 normal_eyespace = (length(prd.normal) > 0.f) ? normalize(normal_matrix * prd.normal) : make_float3(0., 0., 0.);
			normal += normal_eyespace;
			seed = prd.seed;
			depth_map += prd.depthMap;
		} while (--samples_per_pixel);

		//
		// Update the output buffer
		//
		unsigned int spp = sqrt_num_samples*sqrt_num_samples;
		float3 pixel_color = result / spp;
		float3 pixel_normal = normal / (spp);
		float pixel_depth =1- depth_map /6;


		if (frame_number > 1)
		{
			float a = 1.0f / (float)frame_number;
			float3 old_color = make_float3(output_buffer[launch_index]);
			float3 old_normal = make_float3(input_normal_buffer[launch_index]);
			float3 old_depth = make_float3(input_depth_buffer[launch_index]);

			if (lerp_bol) {
				output_buffer[launch_index] = make_float4(lerp(old_color, pixel_color, a), 1.0f);
				// this is not strictly a correct accumulation of normals, but it will do for this sample
				float3 accum_normal = lerp(old_normal, pixel_normal, a);
				input_normal_buffer[launch_index] = make_float4((length(accum_normal) > 0.f) ? normalize(accum_normal) : pixel_normal, 1.0f);
				input_depth_buffer[launch_index] = make_float4(lerp(old_depth, make_float3(pixel_depth), a), 1.0f);

			}
			else {
				output_buffer[launch_index] = make_float4(pixel_color, 1.0f);
				input_normal_buffer[launch_index] = make_float4(pixel_normal, 1.0f);
				input_depth_buffer[launch_index] = make_float4(make_float3(pixel_depth), 1.0f);

			}
		}
		else
		{
			output_buffer[launch_index] = make_float4(pixel_color, 1.0f);
			input_normal_buffer[launch_index] = make_float4(pixel_normal, 1.0f);
			input_depth_buffer[launch_index] = make_float4(make_float3(pixel_depth), 1.0f);


		}

	
}


//-----------------------------------------------------------------------------
//
//  Emissive surface closest-hit
//
//-----------------------------------------------------------------------------



RT_PROGRAM void diffuseEmitter()
{

	//current_prd.radiance = current_prd.countEmitted ? emission_color : make_float3(1.f);
	current_prd.done = true;

	//// TODO: Find out what the albedo buffer should really have. For now just set to white for 
	//// light sources.
	//if (current_prd.depth == 0 && length(current_prd.normal) == 0)
	//{

	//	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	//	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	//	float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	//	current_prd.normal = ffnormal;
	//	current_prd.depthMap =t_hit;
	//}


}


//-----------------------------------------------------------------------------
//
//  Lambertian surface closest-hit
//
//-----------------------------------------------------------------------------

RT_PROGRAM void alpha_texture_hit()
{
	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	float3 hitpoint = ray.origin + t_hit * ray.direction;
	//float distance = length(hitpoint - ray.origin);
	//current_prd.alph = false;
	//
	// Generate a reflection ray.  This will be traced back in ray-gen.
	//
	float3 direction = ray.direction;
	current_prd.origin = hitpoint;


	const float3 Kd_val = make_float3(tex2D(Kd_map, texcoord.x, texcoord.y));
	const float3 Ks_val = make_float3(tex2D(Ks_map, texcoord.x, texcoord.y));
	const float3 d_val = make_float3(tex2D(d_map, texcoord.x, texcoord.y));
	const float3 bump_val2 = make_float3(tex2D(bump_map, texcoord.x, texcoord.y));
	const float3 bump_val = make_float3(bump_val2.z, bump_val2.z, bump_val2.z);
	// Initialze per-ray data


	// NOTE: f/pdf = 1 since we are perfectly importance sampling lambertian
	// with cosine density.
	if (d_val.x == 0) {// hit leaves texture ,trace again

		
		Ray ray2 = make_Ray(hitpoint, direction, 0u, scene_epsilon, RT_DEFAULT_MAX);
		rtTrace(top_object, ray2, current_prd);
	
		//current_prd.alpha = true;
		if (current_prd.depth == 0 )
		{				
			current_prd.depthMap += t_hit;			
		}
		
		
	}
	else
	{		
		if (current_prd.depth == 0)
		{
			if (!current_prd.alpha) 
			{
				current_prd.normal = ffnormal;
			}
			current_prd.depthMap = t_hit;
		}



		float z1 = rnd(current_prd.seed);
		float z2 = rnd(current_prd.seed);
		float3 p;
		cosine_sample_hemisphere(z1, z2, p);
		optix::Onb onb(ffnormal);
		onb.inverse_transform(p);
		current_prd.direction = p;


		current_prd.attenuation = current_prd.attenuation/**Ka*/*Kd_val *Ks_val*bump_val;
		current_prd.countEmitted = false;
		
		// Next event estimation (compute direct lighting).
		//
		unsigned int num_lights = lights.size();
		float3 result =  Kd_val*Ka*Ks_val*bump_val;//make_float3(0.0);

		for (int i = 0; i < num_lights; ++i)
		{
			// Choose random point on light
			ParallelogramLight light = lights[i];
			//const float z1 = rnd(current_prd.seed);
			//const float z2 = rnd(current_prd.seed);
			const float3 light_pos = light.corner;//+light.v1 * z1 + light.v2 * z2;

			// Calculate properties of light sample (for area based pdf)
			const float  Ldist = length(light_pos - hitpoint);
			const float3 L = normalize(light_pos - hitpoint);
			const float  nDl = dot(ffnormal, L);
			//const float  LnDl = dot(light.normal, L);
			
			// cast shadow ray
			if (nDl > 0.0f /*&& LnDl > 0.0f*/)
			{
				PerRayData_pathtrace_shadow shadow_prd;
				shadow_prd.inShadow = false;
				// Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
				Ray shadow_ray = make_Ray(hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon);
				rtTrace(top_object, shadow_ray, shadow_prd);

				if (!shadow_prd.inShadow)
				{
					//const float A = length(cross(light.v1, light.v2));
					// convert area based pdf to solid angle
					const float weight = nDl /** LnDl * A *// (M_PIf * Ldist * Ldist);
					result += light.emission * weight ;
				}
			}
		}

		current_prd.radiance = result;
	}
}


RT_PROGRAM void texture_hit()
{

	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);


	float3 hitpoint = ray.origin + t_hit * ray.direction;
	

	if (current_prd.depth == 0 )
	{
		if (!current_prd.alpha) {
			current_prd.normal = ffnormal;
		}
		current_prd.depthMap = t_hit;
		
	}
	
	//
	// Generate a reflection ray.  This will be traced back in ray-gen.
	//
	current_prd.origin = hitpoint;

	float z1 = rnd(current_prd.seed);
	float z2 = rnd(current_prd.seed);
	float3 p;
	cosine_sample_hemisphere(z1, z2, p);
	optix::Onb onb(ffnormal);
	onb.inverse_transform(p);
	current_prd.direction = p;

	const float3 Kd_val = make_float3(tex2D(Kd_map, texcoord.x, texcoord.y));
	const float3 Ks_val = make_float3(tex2D(Ks_map, texcoord.x, texcoord.y));
	const float3 bump_val2 = make_float3(tex2D(bump_map, texcoord.x, texcoord.y));
	const float3 bump_val = make_float3(bump_val2.z, bump_val2.z, bump_val2.z);


	// NOTE: f/pdf = 1 since we are perfectly importance sampling lambertian
	// with cosine density.
	current_prd.attenuation = current_prd.attenuation*Kd_val*Ks_val*bump_val;
	current_prd.countEmitted = false;

	//
	// Next event estimation (compute direct lighting).
	//
	unsigned int num_lights = lights.size();
	float3 result =  Kd_val*Ka*Ks_val*bump_val;//make_float3(0.0);

	for (int i = 0; i < num_lights; ++i)
	{
		// Choose random point on light
		ParallelogramLight light = lights[i];
		const float z1 = rnd(current_prd.seed);
		const float z2 = rnd(current_prd.seed);
		const float3 light_pos = light.corner;// +light.v1 * z1 + light.v2 * z2;//

		// Calculate properties of light sample (for area based pdf)
		const float  Ldist = length(light_pos - hitpoint);
		const float3 L = normalize(light_pos - hitpoint);
		const float  nDl = dot(ffnormal, L);
		//const float  LnDl = dot(light.normal, L);

		// cast shadow ray
		if (nDl > 0.0f/* && LnDl > 0.0f*/)
		{
			PerRayData_pathtrace_shadow shadow_prd;
			shadow_prd.inShadow = false;
			// Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
			Ray shadow_ray = make_Ray(hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon);
			rtTrace(top_object, shadow_ray, shadow_prd);

			if (!shadow_prd.inShadow)
			{
				//const float A = length(cross(light.v1, light.v2));
				//convert area based pdf to solid angle
				const float weight = nDl /** LnDl * A *// (M_PIf * Ldist * Ldist);
				result += light.emission * weight ;
			}
			
		}
	}

	current_prd.radiance = result;
}

RT_PROGRAM void mirror()
{
	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	float3 hitpoint = ray.origin + t_hit * ray.direction;


	if (current_prd.depth == 0 )
	{
		if (!current_prd.alpha) {
			current_prd.normal = ffnormal;
		}
		current_prd.depthMap = t_hit;
	}

	//
	// Generate a reflection ray.  This will be traced back in ray-gen.
	//
	current_prd.origin = hitpoint;


	float3 p;
	p = reflect(ray.direction, ffnormal);
	//optix::Onb onb(ffnormal);
	//onb.inverse_transform(p);
	current_prd.direction = p;

	// NOTE: f/pdf = 1 since we are perfectly importance sampling lambertian
	// with cosine density.
	current_prd.attenuation = current_prd.attenuation * diffuse_color+(specular_color*0.3);//
	current_prd.countEmitted = false;

	// Next event estimation (compute direct lighting).
	//
	unsigned int num_lights = lights.size();
	float3 result = diffuse_color*Ka + specular_color*0.3;

	for (int i = 0; i < num_lights; ++i)
	{
		// Choose random point on light
		ParallelogramLight light = lights[i];
		const float z1 = rnd(current_prd.seed);
		const float z2 = rnd(current_prd.seed);
		const float3 light_pos = light.corner +light.v1 * z1 + light.v2 * z2;//

											  // Calculate properties of light sample (for area based pdf)
		const float  Ldist = length(light_pos - hitpoint);
		const float3 L = normalize(light_pos - hitpoint);
		const float  nDl = dot(ffnormal, L);		// the normal of hitpoint's normal dot L to hitpoint.  
		const float  LnDl = dot(light.normal, L);	// the normal of L's		normal dot L to hitpoint. 

													// cast shadow ray
		if (nDl > 0.0f && LnDl > 0.0f)
		{
			PerRayData_pathtrace_shadow shadow_prd;
			shadow_prd.inShadow = false;
			// Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
			Ray shadow_ray = make_Ray(hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon);
			rtTrace(top_object, shadow_ray, shadow_prd);

			if (!shadow_prd.inShadow)
			{

				const float A = length(cross(light.v1, light.v2));
				// convert area based pdf to solid angle
				const float weight = nDl  *LnDl * A  / (M_PIf * Ldist * Ldist);
				result += light.emission * weight;
			}
		}
	}

	current_prd.radiance = result;
}



RT_PROGRAM void diffuse()
{
	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	float3 hitpoint = ray.origin + t_hit * ray.direction;
	

	if (current_prd.depth == 0 )
	{
		if (!current_prd.alpha) {
			current_prd.normal = ffnormal;
		}
		current_prd.depthMap = t_hit;
	}
	//
	// Generate a reflection ray.  This will be traced back in ray-gen.
	//
	current_prd.origin = hitpoint;

	float z1 = rnd(current_prd.seed);
	float z2 = rnd(current_prd.seed);
	float3 p;
	cosine_sample_hemisphere(z1, z2, p);
	optix::Onb onb(ffnormal);
	onb.inverse_transform(p);
	current_prd.direction = p;

	// NOTE: f/pdf = 1 since we are perfectly importance sampling lambertian
	// with cosine density.
	current_prd.attenuation = current_prd.attenuation * diffuse_color+(specular_color*0.3);//
	current_prd.countEmitted = false;

	// Next event estimation (compute direct lighting).
	//
	unsigned int num_lights = lights.size();
	float3 result = diffuse_color *Ka+specular_color*0.3; //make_float3(0.0);//;//;// 

	for (int i = 0; i < num_lights; ++i)
	{
		// Choose random point on light
		ParallelogramLight light = lights[i];
		//const float z1 = rnd(current_prd.seed);
		//const float z2 = rnd(current_prd.seed);
		const float3 light_pos = light.corner;// +light.v1 * z1 + light.v2 * z2;

		// Calculate properties of light sample (for area based pdf)
		const float  Ldist = length(light_pos - hitpoint);
		const float3 L = normalize(light_pos - hitpoint);
		const float  nDl = dot(ffnormal, L);		// the normal of hitpoint's normal dot L to hitpoint.  
		//const float  LnDl = dot(light.normal, L);	// the normal of L's		normal dot L to hitpoint. 

		// cast shadow ray
		if (nDl > 0.0f/* && LnDl > 0.0f*/)
		{
			PerRayData_pathtrace_shadow shadow_prd;
			shadow_prd.inShadow = false;
			// Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
			Ray shadow_ray = make_Ray(hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon);
			rtTrace(top_object, shadow_ray, shadow_prd);

			if (!shadow_prd.inShadow)
			{

				//const float A = length(cross(light.v1, light.v2));
				// convert area based pdf to solid angle
				const float weight = nDl /* *LnDl * A*/ / (M_PIf * Ldist * Ldist);
				result += light.emission * weight ;
			}
		}
	}

	current_prd.radiance = result;
}


//-----------------------------------------------------------------------------
//
//  Shadow any-hit
//
//-----------------------------------------------------------------------------


RT_PROGRAM void shadow()
{

	const float3 d_val = make_float3(tex2D(d_map, texcoord.x, texcoord.y));
	if (d_val.x != 0) {
		current_prd_shadow.inShadow = true;
		rtTerminateRay();
	}

}



//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
	
	printf("%d\n", rtGetExceptionCode());
	output_buffer[launch_index] = make_float4(bad_color, 1.0f);
	input_normal_buffer[launch_index] = make_float4(bad_color, 1.0f);
	input_depth_buffer[launch_index] = make_float4(bad_color, 1.0f);
}


//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------


RT_PROGRAM void miss()
{
	
	current_prd.radiance = bg_color;
	current_prd.done = true;

	// TODO: Find out what the albedo buffer should really have. For now just set to black for misses.
	if (current_prd.depth == 0)
	{
		current_prd.normal = make_float3(0, 0, 0);
		current_prd.depthMap = 0;
	}

}

rtTextureSampler<float4, 2>		envmap;
RT_PROGRAM void envmap_miss()
{
	
	float theta = atan2f(ray.direction.x, ray.direction.z);
	float phi = M_PIf * 0.5f - acosf(ray.direction.y);
	float u = (theta + M_PIf) * (0.5f * M_1_PIf);
	float v = 0.5f * (1.0f + sin(phi));
	float3 result = make_float3(tex2D(envmap, u, v));
	
	current_prd.radiance = result;
	//current_prd.attenuation =make_float3(0.0);
		//make_float3(tex2D(envmap, u, v));
	current_prd.done = true;

	if (current_prd.depth == 0)
	{
		current_prd.normal = make_float3(0, 0, 0);
		current_prd.depthMap = 0;
	}

}